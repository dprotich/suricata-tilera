
#include <hip/hip_runtime.h>
/* Copyright (C) 2007-2010 Open Information Security Foundation
 *
 * You can copy, redistribute or modify this Program under the terms of
 * the GNU General Public License version 2 as published by the Free
 * Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * version 2 along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA
 * 02110-1301, USA.
 */

/**
 * \file
 *
 * \author Anoop Saldanha <anoopsaldanha@gmail.com>
 *
 * The Cuda kernel for MPM B2G.
 *
 * \todo This is a basic version of the kernel.  Modify it to support multiple
 *       blocks of threads.  Make use of shared memory/texture memory.
 */

#define B2G_CUDA_Q 2
#define CUDA_THREADS 4000
#define B2G_CUDA_HASHSHIFT 4
#define B2G_CUDA_TYPE unsigned int
#define B2G_CUDA_HASH16(a, b) (((a) << B2G_CUDA_HASHSHIFT) | (b))
#define u8_tolower(c) g_u8_lowercasetable[(c)]

typedef struct SCCudaPBPacketDataForGPU_ {
    /* holds the value B2gCtx->m */
    unsigned int m;
    /* holds B2gCtx->B2g */
    unsigned int table;
    /* holds the length of the payload */
    unsigned int payload_len;
    /* holds the payload */
    unsigned char payload;
} SCCudaPBPacketDataForGPU;

extern "C"
__global__ void B2gCudaSearchBNDMq(unsigned short *results_buffer,
                                   unsigned char *packets_buffer,
                                   unsigned int *packets_offset_buffer,
                                   unsigned int *packets_payload_offset_buffer,
                                   unsigned int nop,
                                   unsigned char *g_u8_lowercasetable)
 {
    unsigned int tid = blockIdx.x * 32 + threadIdx.x;
    /* if the thread id is greater than the no of packets sent in the packets
     * buffer, terminate the thread */
    //if (tid <= nop)
    if (tid >= nop)
        return;

    SCCudaPBPacketDataForGPU *packet = (SCCudaPBPacketDataForGPU *)(packets_buffer + packets_offset_buffer[tid]);
    unsigned int m = packet->m;
    unsigned char *buf = &packet->payload;
    unsigned int buflen = packet->payload_len;
    unsigned int *B2G = (unsigned int *)packet->table;
    unsigned int pos = m - B2G_CUDA_Q + 1;
    B2G_CUDA_TYPE d;
    unsigned short h;
    unsigned int first;
    unsigned int j = 0;

    unsigned short *matches_count = results_buffer + packets_payload_offset_buffer[tid] + tid;
    //unsigned short *matches_count = results_buffer + packets_payload_offset_buffer[1] + 1;
    //unsigned short *offsets = results_buffer + packets_payload_offset_buffer[1] + 1 + 1;
    unsigned short *offsets = matches_count + 1;
    // temporarily hold the results here, before we shift it to matches_count
    // before returning
    unsigned short matches = 0;

    while (pos <= (buflen - B2G_CUDA_Q + 1)) {
        h = B2G_CUDA_HASH16(u8_tolower(buf[pos - 1]), u8_tolower(buf[pos]));
        d = B2G[h];

        if (d != 0) {
            j = pos;
            first = pos - (m - B2G_CUDA_Q + 1);

            do {
                j = j - 1;
                if (d >= (1 << (m - 1))) {
                    if (j > first) {
                        pos = j;
                    } else {
                        offsets[matches++] = j;
                    }
                }

                if (j == 0)
                    break;

                h = B2G_CUDA_HASH16(u8_tolower(buf[j - 1]), u8_tolower(buf[j]));
                d = (d << 1) & B2G[h];
            } while (d != 0);
        }
        pos = pos + m - B2G_CUDA_Q + 1;
    }

    matches_count[0] = matches;

    return;
}
